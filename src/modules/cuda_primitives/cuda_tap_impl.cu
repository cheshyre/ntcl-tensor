#include "hip/hip_runtime.h"
#include <complex>
#include <iostream>
#include <hip/hip_complex.h>

#include <cuda_common.h>

// Add
template <typename T>
__global__ void cuda_tap_add_kernel(T *dst, T *src, uint64_t ndim, float alpha) {
  uint64_t idx = ((uint64_t) blockIdx.x)*((uint64_t) blockDim.x) + threadIdx.x;

  if ( idx < ndim ) dst[idx] += alpha*src[idx];
}

__global__ void cuda_tap_add_kernel_c64(hipComplex *dst, hipComplex *src, uint64_t ndim, hipComplex alpha) {
  uint64_t idx = ((uint64_t) blockIdx.x)*((uint64_t) blockDim.x) + threadIdx.x;

  if ( idx < ndim ) dst[idx] = hipCaddf(dst[idx], hipCmulf(alpha, src[idx]));
}

__global__ void cuda_tap_add_kernel_c128(hipDoubleComplex *dst, hipDoubleComplex *src, uint64_t ndim, hipDoubleComplex alpha) {
  uint64_t idx = ((uint64_t) blockIdx.x)*((uint64_t) blockDim.x) + threadIdx.x;

  if ( idx < ndim ) dst[idx] = hipCadd(dst[idx], hipCmul(alpha, src[idx]));
}

extern "C" void cuda_tap_add_real32(float *dst, float*src, int64_t ndim, float alpha, hipStream_t *stream) {
  int block_size = 256;
  int nblocks = (int) (ndim-1)/(block_size)+1;
  dim3 blockGrid(nblocks);
  dim3 thread_per_block(block_size);
  uint64_t nelements = ndim;

  if ( stream ) {
    cuda_tap_add_kernel<float><<<nblocks, block_size, 0, *stream>>>(dst, src, nelements, alpha);
  } else {
    cuda_tap_add_kernel<float><<<nblocks, block_size>>>(dst, src, nelements, alpha);
  }
}

extern "C" void cuda_tap_add_real64(double *dst, double*src, int64_t ndim, double alpha, hipStream_t *stream) {
  int block_size = 256;
  int nblocks = (int) (ndim-1)/(block_size)+1;
  dim3 blockGrid(nblocks);
  dim3 thread_per_block(block_size);
  uint64_t nelements = ndim;

  if ( stream ) {
    cuda_tap_add_kernel<double><<<nblocks, block_size, 0, *stream>>>(dst, src, nelements, alpha);
  } else {
    cuda_tap_add_kernel<double><<<nblocks, block_size>>>(dst, src, nelements, alpha);
  }
}

extern "C" void cuda_tap_add_complex64(hipComplex *dst, hipComplex *src, int64_t ndim, hipComplex alpha, hipStream_t *stream) {

  int block_size = 256;
  int nblocks = (int) (ndim-1)/(block_size)+1;
  dim3 blockGrid(nblocks);
  dim3 thread_per_block(block_size);
  uint64_t nelements = ndim;

  if ( stream ) {
    cuda_tap_add_kernel_c64<<<nblocks, block_size, 0, *stream>>>(dst, src, nelements, alpha);
  } else {
    cuda_tap_add_kernel_c64<<<nblocks, block_size>>>(dst, src, nelements, alpha);
  }
}

extern "C" void cuda_tap_add_complex128(hipDoubleComplex *dst, hipDoubleComplex*src,
    int64_t ndim, hipDoubleComplex alpha, hipStream_t *stream) {

  int block_size = 256;
  int nblocks = (int) (ndim-1)/(block_size)+1;
  dim3 blockGrid(nblocks);
  dim3 thread_per_block(block_size);
  uint64_t nelements = ndim;

  if ( stream ) {
    cuda_tap_add_kernel_c128<<<nblocks, block_size, 0, *stream>>>(dst, src, nelements, alpha);
  } else {
    cuda_tap_add_kernel_c128<<<nblocks, block_size>>>(dst, src, nelements, alpha);
  }
}

// Multiply
template <typename T>
__global__ void cuda_tap_multiply_kernel(T *dst, T *src, uint64_t ndim, float alpha) {
  uint64_t idx = ((uint64_t) blockIdx.x)*((uint64_t) blockDim.x) + threadIdx.x;

  if ( idx < ndim ) dst[idx] *= alpha*src[idx];
}

__global__ void cuda_tap_multiply_kernel_c64(hipComplex *dst, hipComplex *src, uint64_t ndim, hipComplex alpha) {
  uint64_t idx = ((uint64_t) blockIdx.x)*((uint64_t) blockDim.x) + threadIdx.x;

  if ( idx < ndim ) dst[idx] = hipCmulf(dst[idx], hipCmulf(alpha, src[idx]));
}

__global__ void cuda_tap_multiply_kernel_c128(hipDoubleComplex *dst, hipDoubleComplex *src, uint64_t ndim, hipDoubleComplex alpha) {
  uint64_t idx = ((uint64_t) blockIdx.x)*((uint64_t) blockDim.x) + threadIdx.x;

  if ( idx < ndim ) dst[idx] = hipCmul(dst[idx], hipCmul(alpha, src[idx]));
}

extern "C" void cuda_tap_multiply_real32(float *dst, float*src, int64_t ndim, float alpha, hipStream_t *stream) {
  int block_size = 256;
  int nblocks = (int) (ndim-1)/(block_size)+1;
  dim3 blockGrid(nblocks);
  dim3 thread_per_block(block_size);
  uint64_t nelements = ndim;

  if ( stream ) {
    cuda_tap_multiply_kernel<float><<<nblocks, block_size, 0, *stream>>>(dst, src, nelements, alpha);
  } else {
    cuda_tap_multiply_kernel<float><<<nblocks, block_size>>>(dst, src, nelements, alpha);
  }
}

extern "C" void cuda_tap_multiply_real64(double *dst, double*src, int64_t ndim, double alpha, hipStream_t *stream) {
  int block_size = 256;
  int nblocks = (int) (ndim-1)/(block_size)+1;
  dim3 blockGrid(nblocks);
  dim3 thread_per_block(block_size);
  uint64_t nelements = ndim;

  if ( stream ) {
    cuda_tap_multiply_kernel<double><<<nblocks, block_size, 0, *stream>>>(dst, src, nelements, alpha);
  } else {
    cuda_tap_multiply_kernel<double><<<nblocks, block_size>>>(dst, src, nelements, alpha);
  }
}

extern "C" void cuda_tap_multiply_complex64(hipComplex *dst, hipComplex *src, int64_t ndim, hipComplex alpha, hipStream_t *stream) {

  int block_size = 256;
  int nblocks = (int) (ndim-1)/(block_size)+1;
  dim3 blockGrid(nblocks);
  dim3 thread_per_block(block_size);
  uint64_t nelements = ndim;

  if ( stream ) {
    cuda_tap_multiply_kernel_c64<<<nblocks, block_size, 0, *stream>>>(dst, src, nelements, alpha);
  } else {
    cuda_tap_multiply_kernel_c64<<<nblocks, block_size>>>(dst, src, nelements, alpha);
  }
}

extern "C" void cuda_tap_multiply_complex128(hipDoubleComplex *dst, hipDoubleComplex*src,
    int64_t ndim, hipDoubleComplex alpha, hipStream_t *stream) {

  int block_size = 256;
  int nblocks = (int) (ndim-1)/(block_size)+1;
  dim3 blockGrid(nblocks);
  dim3 thread_per_block(block_size);
  uint64_t nelements = ndim;

  if ( stream ) {
    cuda_tap_multiply_kernel_c128<<<nblocks, block_size, 0, *stream>>>(dst, src, nelements, alpha);
  } else {
    cuda_tap_multiply_kernel_c128<<<nblocks, block_size>>>(dst, src, nelements, alpha);
  }
}

// Divide
template <typename T>
__global__ void cuda_tap_divide_kernel(T *dst, T *src, uint64_t ndim, float alpha) {
  uint64_t idx = ((uint64_t) blockIdx.x)*((uint64_t) blockDim.x) + threadIdx.x;

  if ( idx < ndim ) dst[idx] = dst[idx]*alpha/src[idx];
}

__global__ void cuda_tap_divide_kernel_c64(hipComplex *dst, hipComplex *src, uint64_t ndim, hipComplex alpha) {
  uint64_t idx = ((uint64_t) blockIdx.x)*((uint64_t) blockDim.x) + threadIdx.x;

  if ( idx < ndim ) dst[idx] = hipCmulf(dst[idx], hipCdivf(alpha, src[idx]));
}

__global__ void cuda_tap_divide_kernel_c128(hipDoubleComplex *dst, hipDoubleComplex *src, uint64_t ndim, hipDoubleComplex alpha) {
  uint64_t idx = ((uint64_t) blockIdx.x)*((uint64_t) blockDim.x) + threadIdx.x;

  if ( idx < ndim ) dst[idx] = hipCmul(dst[idx], hipCdiv(alpha, src[idx]));
}

extern "C" void cuda_tap_divide_real32(float *dst, float*src, int64_t ndim, float alpha, hipStream_t *stream) {
  int block_size = 256;
  int nblocks = (int) (ndim-1)/(block_size)+1;
  dim3 blockGrid(nblocks);
  dim3 thread_per_block(block_size);
  uint64_t nelements = ndim;

  if ( stream ) {
    cuda_tap_divide_kernel<float><<<nblocks, block_size, 0, *stream>>>(dst, src, nelements, alpha);
  } else {
    cuda_tap_divide_kernel<float><<<nblocks, block_size>>>(dst, src, nelements, alpha);
  }
}

extern "C" void cuda_tap_divide_real64(double *dst, double*src, int64_t ndim, double alpha, hipStream_t *stream) {
  int block_size = 256;
  int nblocks = (int) (ndim-1)/(block_size)+1;
  dim3 blockGrid(nblocks);
  dim3 thread_per_block(block_size);
  uint64_t nelements = ndim;

  if ( stream ) {
    cuda_tap_divide_kernel<double><<<nblocks, block_size, 0, *stream>>>(dst, src, nelements, alpha);
  } else {
    cuda_tap_divide_kernel<double><<<nblocks, block_size>>>(dst, src, nelements, alpha);
  }
}

extern "C" void cuda_tap_divide_complex64(hipComplex *dst, hipComplex *src, int64_t ndim, hipComplex alpha, hipStream_t *stream) {

  int block_size = 256;
  int nblocks = (int) (ndim-1)/(block_size)+1;
  dim3 blockGrid(nblocks);
  dim3 thread_per_block(block_size);
  uint64_t nelements = ndim;

  if ( stream ) {
    cuda_tap_divide_kernel_c64<<<nblocks, block_size, 0, *stream>>>(dst, src, nelements, alpha);
  } else {
    cuda_tap_divide_kernel_c64<<<nblocks, block_size>>>(dst, src, nelements, alpha);
  }
}

extern "C" void cuda_tap_divide_complex128(hipDoubleComplex *dst, hipDoubleComplex*src,
    int64_t ndim, hipDoubleComplex alpha, hipStream_t *stream) {

  int block_size = 256;
  int nblocks = (int) (ndim-1)/(block_size)+1;
  dim3 blockGrid(nblocks);
  dim3 thread_per_block(block_size);
  uint64_t nelements = ndim;

  if ( stream ) {
    cuda_tap_divide_kernel_c128<<<nblocks, block_size, 0, *stream>>>(dst, src, nelements, alpha);
  } else {
    cuda_tap_divide_kernel_c128<<<nblocks, block_size>>>(dst, src, nelements, alpha);
  }
}

// Scalar add
template <typename T>
__global__ void cuda_tap_scalar_add_kernel(T *dst, uint64_t ndim, float alpha) {
  uint64_t idx = ((uint64_t) blockIdx.x)*((uint64_t) blockDim.x) + threadIdx.x;

  if ( idx < ndim ) dst[idx] += alpha;
}

__global__ void cuda_tap_scalar_add_kernel_c64(hipComplex *dst, uint64_t ndim, hipComplex alpha) {
  uint64_t idx = ((uint64_t) blockIdx.x)*((uint64_t) blockDim.x) + threadIdx.x;

  if ( idx < ndim ) dst[idx] = hipCaddf(dst[idx], alpha);
}

__global__ void cuda_tap_scalar_add_kernel_c128(hipDoubleComplex *dst, uint64_t ndim, hipDoubleComplex alpha) {
  uint64_t idx = ((uint64_t) blockIdx.x)*((uint64_t) blockDim.x) + threadIdx.x;

  if ( idx < ndim ) dst[idx] = hipCadd(dst[idx], alpha);
}

extern "C" void cuda_tap_scalar_add_real32(float *dst, int64_t ndim, float alpha, hipStream_t *stream) {
  int block_size = 256;
  int nblocks = (int) (ndim-1)/(block_size)+1;
  dim3 blockGrid(nblocks);
  dim3 thread_per_block(block_size);
  uint64_t nelements = ndim;

  if ( stream ) {
    cuda_tap_scalar_add_kernel<float><<<nblocks, block_size, 0, *stream>>>(dst, nelements, alpha);
  } else {
    cuda_tap_scalar_add_kernel<float><<<nblocks, block_size>>>(dst, nelements, alpha);
  }
}

extern "C" void cuda_tap_scalar_add_real64(double *dst, int64_t ndim, double alpha, hipStream_t *stream) {
  int block_size = 256;
  int nblocks = (int) (ndim-1)/(block_size)+1;
  dim3 blockGrid(nblocks);
  dim3 thread_per_block(block_size);
  uint64_t nelements = ndim;

  if ( stream ) {
    cuda_tap_scalar_add_kernel<double><<<nblocks, block_size, 0, *stream>>>(dst, nelements, alpha);
  } else {
    cuda_tap_scalar_add_kernel<double><<<nblocks, block_size>>>(dst, nelements, alpha);
  }
}

extern "C" void cuda_tap_scalar_add_complex64(hipComplex *dst, int64_t ndim, hipComplex alpha, hipStream_t *stream) {

  int block_size = 256;
  int nblocks = (int) (ndim-1)/(block_size)+1;
  dim3 blockGrid(nblocks);
  dim3 thread_per_block(block_size);
  uint64_t nelements = ndim;

  if ( stream ) {
    cuda_tap_scalar_add_kernel_c64<<<nblocks, block_size, 0, *stream>>>(dst, nelements, alpha);
  } else {
    cuda_tap_scalar_add_kernel_c64<<<nblocks, block_size>>>(dst, nelements, alpha);
  }
}

extern "C" void cuda_tap_scalar_add_complex128(hipDoubleComplex *dst,
    int64_t ndim, hipDoubleComplex alpha, hipStream_t *stream) {

  int block_size = 256;
  int nblocks = (int) (ndim-1)/(block_size)+1;
  dim3 blockGrid(nblocks);
  dim3 thread_per_block(block_size);
  uint64_t nelements = ndim;

  if ( stream ) {
    cuda_tap_scalar_add_kernel_c128<<<nblocks, block_size, 0, *stream>>>(dst, nelements, alpha);
  } else {
    cuda_tap_scalar_add_kernel_c128<<<nblocks, block_size>>>(dst, nelements, alpha);
  }
}

// Scalar multiply
template <typename T>
__global__ void cuda_tap_scalar_multiply_kernel(T *dst, uint64_t ndim, float alpha) {
  uint64_t idx = ((uint64_t) blockIdx.x)*((uint64_t) blockDim.x) + threadIdx.x;

  if ( idx < ndim ) dst[idx] *= alpha;
}

__global__ void cuda_tap_scalar_multiply_kernel_c64(hipComplex *dst, uint64_t ndim, hipComplex alpha) {
  uint64_t idx = ((uint64_t) blockIdx.x)*((uint64_t) blockDim.x) + threadIdx.x;

  if ( idx < ndim ) dst[idx] = hipCmulf(dst[idx], alpha);
}

__global__ void cuda_tap_scalar_multiply_kernel_c128(hipDoubleComplex *dst, uint64_t ndim, hipDoubleComplex alpha) {
  uint64_t idx = ((uint64_t) blockIdx.x)*((uint64_t) blockDim.x) + threadIdx.x;

  if ( idx < ndim ) dst[idx] = hipCmul(dst[idx], alpha);
}

extern "C" void cuda_tap_scalar_multiply_real32(float *dst, int64_t ndim, float alpha, hipStream_t *stream) {
  int block_size = 256;
  int nblocks = (int) (ndim-1)/(block_size)+1;
  dim3 blockGrid(nblocks);
  dim3 thread_per_block(block_size);
  uint64_t nelements = ndim;

  if ( stream ) {
    cuda_tap_scalar_multiply_kernel<float><<<nblocks, block_size, 0, *stream>>>(dst, nelements, alpha);
  } else {
    cuda_tap_scalar_multiply_kernel<float><<<nblocks, block_size>>>(dst, nelements, alpha);
  }
}

extern "C" void cuda_tap_scalar_multiply_real64(double *dst, int64_t ndim, double alpha, hipStream_t *stream) {
  int block_size = 256;
  int nblocks = (int) (ndim-1)/(block_size)+1;
  dim3 blockGrid(nblocks);
  dim3 thread_per_block(block_size);
  uint64_t nelements = ndim;

  if ( stream ) {
    cuda_tap_scalar_multiply_kernel<double><<<nblocks, block_size, 0, *stream>>>(dst, nelements, alpha);
  } else {
    cuda_tap_scalar_multiply_kernel<double><<<nblocks, block_size>>>(dst, nelements, alpha);
  }
}

extern "C" void cuda_tap_scalar_multiply_complex64(hipComplex *dst, int64_t ndim, hipComplex alpha, hipStream_t *stream) {

  int block_size = 256;
  int nblocks = (int) (ndim-1)/(block_size)+1;
  dim3 blockGrid(nblocks);
  dim3 thread_per_block(block_size);
  uint64_t nelements = ndim;

  if ( stream ) {
    cuda_tap_scalar_multiply_kernel_c64<<<nblocks, block_size, 0, *stream>>>(dst, nelements, alpha);
  } else {
    cuda_tap_scalar_multiply_kernel_c64<<<nblocks, block_size>>>(dst, nelements, alpha);
  }
}

extern "C" void cuda_tap_scalar_multiply_complex128(hipDoubleComplex *dst,
    int64_t ndim, hipDoubleComplex alpha, hipStream_t *stream) {

  int block_size = 256;
  int nblocks = (int) (ndim-1)/(block_size)+1;
  dim3 blockGrid(nblocks);
  dim3 thread_per_block(block_size);
  uint64_t nelements = ndim;

  if ( stream ) {
    cuda_tap_scalar_multiply_kernel_c128<<<nblocks, block_size, 0, *stream>>>(dst, nelements, alpha);
  } else {
    cuda_tap_scalar_multiply_kernel_c128<<<nblocks, block_size>>>(dst, nelements, alpha);
  }
}

// Scalar inverse
template <typename T>
__global__ void cuda_tap_scalar_inverse_kernel(T *dst, uint64_t ndim, float alpha) {
  uint64_t idx = ((uint64_t) blockIdx.x)*((uint64_t) blockDim.x) + threadIdx.x;

  if ( idx < ndim ) dst[idx] = alpha/dst[idx];
}

__global__ void cuda_tap_scalar_inverse_kernel_c64(hipComplex *dst, uint64_t ndim, hipComplex alpha) {
  uint64_t idx = ((uint64_t) blockIdx.x)*((uint64_t) blockDim.x) + threadIdx.x;

  if ( idx < ndim ) dst[idx] = hipCdivf(alpha, dst[idx]);
}

__global__ void cuda_tap_scalar_inverse_kernel_c128(hipDoubleComplex *dst, uint64_t ndim, hipDoubleComplex alpha) {
  uint64_t idx = ((uint64_t) blockIdx.x)*((uint64_t) blockDim.x) + threadIdx.x;

  if ( idx < ndim ) dst[idx] = hipCdiv(alpha, dst[idx]);
}

extern "C" void cuda_tap_scalar_inverse_real32(float *dst, int64_t ndim, float alpha, hipStream_t *stream) {
  int block_size = 256;
  int nblocks = (int) (ndim-1)/(block_size)+1;
  dim3 blockGrid(nblocks);
  dim3 thread_per_block(block_size);
  uint64_t nelements = ndim;

  if ( stream ) {
    cuda_tap_scalar_inverse_kernel<float><<<nblocks, block_size, 0, *stream>>>(dst, nelements, alpha);
  } else {
    cuda_tap_scalar_inverse_kernel<float><<<nblocks, block_size>>>(dst, nelements, alpha);
  }
}

extern "C" void cuda_tap_scalar_inverse_real64(double *dst, int64_t ndim, double alpha, hipStream_t *stream) {
  int block_size = 256;
  int nblocks = (int) (ndim-1)/(block_size)+1;
  dim3 blockGrid(nblocks);
  dim3 thread_per_block(block_size);
  uint64_t nelements = ndim;

  if ( stream ) {
    cuda_tap_scalar_inverse_kernel<double><<<nblocks, block_size, 0, *stream>>>(dst, nelements, alpha);
  } else {
    cuda_tap_scalar_inverse_kernel<double><<<nblocks, block_size>>>(dst, nelements, alpha);
  }
}

extern "C" void cuda_tap_scalar_inverse_complex64(hipComplex *dst, int64_t ndim, hipComplex alpha, hipStream_t *stream) {

  int block_size = 256;
  int nblocks = (int) (ndim-1)/(block_size)+1;
  dim3 blockGrid(nblocks);
  dim3 thread_per_block(block_size);
  uint64_t nelements = ndim;

  if ( stream ) {
    cuda_tap_scalar_inverse_kernel_c64<<<nblocks, block_size, 0, *stream>>>(dst, nelements, alpha);
  } else {
    cuda_tap_scalar_inverse_kernel_c64<<<nblocks, block_size>>>(dst, nelements, alpha);
  }
}

extern "C" void cuda_tap_scalar_inverse_complex128(hipDoubleComplex *dst,
    int64_t ndim, hipDoubleComplex alpha, hipStream_t *stream) {

  int block_size = 256;
  int nblocks = (int) (ndim-1)/(block_size)+1;
  dim3 blockGrid(nblocks);
  dim3 thread_per_block(block_size);
  uint64_t nelements = ndim;

  if ( stream ) {
    cuda_tap_scalar_inverse_kernel_c128<<<nblocks, block_size, 0, *stream>>>(dst, nelements, alpha);
  } else {
    cuda_tap_scalar_inverse_kernel_c128<<<nblocks, block_size>>>(dst, nelements, alpha);
  }
}
